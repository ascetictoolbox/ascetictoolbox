#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "extrae_user_events.h"

// Prototypes
__global__ void helloWorld(char*);

// Host function
int main(int argc, char** argv)
{
	int i;

	// desired output
	char str[] = "Hello World!";

	Extrae_init();

	for(i = 0; i < 12; i++)
		str[i] -= i;

	// allocate memory on the device
	char *d_str;
	size_t size = sizeof(str);
	hipMalloc((void**)&d_str, size);

	// copy the string to the device
	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

	// set the grid and block sizes
	dim3 dimGrid(2);   // one block per word
	dim3 dimBlock(6); // one thread per character

	// invoke the kernel
	helloWorld<<< dimGrid, dimBlock >>>(d_str);

	// retrieve the results from the device
	hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

	// Syn all threads (this is optional) HSG
	hipDeviceSynchronize();

	// free up the allocated memory on the device
	hipFree(d_str);

	// everyone's favorite part
	printf("%s\n", str);

	Extrae_fini();

	return 0;
}

// Device kernel
__global__ void helloWorld(char* str)
{
	// determine where in the thread grid we are
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// unmangle output
	str[idx] += idx;
}

